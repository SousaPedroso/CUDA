#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

__global__ void initWith(float num, float *a, int N)
{
  int threadId, gridStride;
  threadId = threadIdx.x + blockIdx.x * blockDim.x;
  gridStride = blockDim.x * gridDim.x;

  for(int i = threadId; i < N; i+= gridStride){
    if (i < N) a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  int threadId, gridStride;
  threadId = threadIdx.x + blockIdx.x * blockDim.x;
  gridStride = blockDim.x * gridDim.x;

  for(int i = threadId; i < N; i+=gridStride){
    if (i<N) result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{

  for(int i = 0; i < N; i+=1){
    if(array[i] != target){
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

hipError_t checkCuda(hipError_t result){
    if (result != hipSuccess){
        fprintf(stderr, "CUDA RUNTIME ERROR: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

int main()
{
  int deviceId, processorCount, blocks, threadsPerBlock;
  hipDeviceProp_t props;
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  if (checkCuda(hipMallocManaged(&a, size)) != hipSuccess){
    exit(1);
  }

  if (checkCuda(hipMallocManaged(&b, size)) != hipSuccess){
    exit(1);
  }

  if (checkCuda(hipMallocManaged(&c, size)) != hipSuccess){
    exit(1);
  }

  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&props, deviceId);

  // https://docs.nvidia.com/cuda/cuda-occupancy-calculator/index.html

  // maxBlocksPerMultiProcessor is 12-above CUDA
  processorCount = props.multiProcessorCount;
  blocks = processorCount*2;
  threadsPerBlock = props.maxThreadsPerBlock;

  /*
    Experiments using hipMemPrefetchAsync to understand its impact on
    page-faulting and memory migration
   * Using hipMemPrefetchAsync each time reduced the total time of execution
   * of the program, decreasing from 4285931ns (97.5%) to 122655ns (52.6%)
   * the amount of time used by initWith method, due to not having anymore
   * data migration overhead

  */
  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c, size, deviceId);

  hipError_t addVectorsErr;
  hipStream_t streamA;
  hipStream_t streamB;
  hipStream_t streamC;
  hipStreamCreate(&streamA);
  hipStreamCreate(&streamB);
  hipStreamCreate(&streamC);

  initWith<<<blocks, threadsPerBlock, 0, streamA>>>(3, a, N);
  initWith<<<blocks, threadsPerBlock, 0, streamB>>>(4, b, N);
  initWith<<<blocks, threadsPerBlock, 0, streamC>>>(0, c, N);

  addVectorsInto<<<blocks, threadsPerBlock>>>(c, a, b, N);

  addVectorsErr = hipGetLastError();
  if (checkCuda(addVectorsErr) != hipSuccess){
    exit(1);
  }

  if (checkCuda(hipDeviceSynchronize()) != hipSuccess){
    exit(1);
  }

  hipMemPrefetchAsync(c, size, hipCpuDeviceId); // Prefetch c to CPU

  checkElementsAre(7, c, N);

  if (checkCuda(hipStreamDestroy(streamA)) != hipSuccess){
    exit(1);
  }

  if (checkCuda(hipStreamDestroy(streamB)) != hipSuccess){
    exit(1);
  }

  if (checkCuda(hipStreamDestroy(streamC)) != hipSuccess){
    exit(1);
  }

  if (checkCuda(hipFree(a)) != hipSuccess){
    exit(1);
  }

  if (checkCuda(hipFree(b)) != hipSuccess){
    exit(1);
  }

  if (checkCuda(hipFree(c)) != hipSuccess){
    exit(1);
  }
}
