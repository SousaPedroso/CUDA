
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N) a[i] = initialValue;
}

int main()
{

  int N = 1000;

  int *a;
  size_t size = N * sizeof(int);

  hipMallocManaged(&a, size);

  size_t threads_per_block = 256;

  size_t number_of_blocks = 4;

  int initialValue = 6;

  initializeElementsTo<<<number_of_blocks, threads_per_block>>>(initialValue, a, N);
  hipDeviceSynchronize();

  for (int i = 0; i < N; ++i)
  {
    if(a[i] != initialValue)
    {
      printf("FAILURE: target value: %d\t a[%d]: %d\n", initialValue, i, a[i]);
      hipFree(a);
      exit(1);
    }
  }
  printf("SUCCESS!\n");

  hipFree(a);
}
